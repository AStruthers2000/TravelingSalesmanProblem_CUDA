#include "hip/hip_runtime.h"
//system includes
#include <iostream>

//cuda includes
#include "hip/hip_runtime.h"
#include ""

//our includes
//#include "Utils/HelperFunctions.cu"
//#include "Problem/ProblemInstance.cu"
#include "Algorithms/AntColony.cuh"
#include "ModelParameters.h"
#include <iostream>

using namespace std;

// evaporation rate must be an element of [0, 1]
const int PHEROMONE_EVAPORATION_RATE = 0.01;

void myexit();

// A helper method for handling erros from CUDA calls
void cudaHandleError(hipError_t error) {
    if (error != hipSuccess) {
        cout << "Failed to perform device operation: " << hipGetErrorString(error);
        error = hipDeviceReset();
    }
}
void getSTSPAdjacencyMatrix(double* matrix, string location, int problemSize) {}

void getATSPAdjacencyMatrix(int* matrix, string location, int nullKey) {}

void updatePheromoneMatrix(double* pheromoneMatrix, int pheromoneMatrixSize, int* antHistories, int antHistoriesSize, int numAnts, int iteration, int problemSize){

    // perform pheromone evaporation on every edge
    for(int i = 0; i < problemSize; i++){
        for(int j = 0; j < problemSize; j++){
            pheromoneMatrix[i*problemSize +j ] = pheromoneMatrix[i*problemSize + j] * PHEROMONE_EVAPORATION_RATE;
        }
    }



    // for each ant history
    for(int i = 0; i < antHistoriesSize/(problemSize+1); i++){

        // for each edge traveled
        for(int j = 0; j <= problemSize; j++){
            int startingCity = antHistories[(problemSize + 1) * i + j];
            int endingCity;

            if(j == problemSize){ // loop back around to starting index if at the end of the path
                endingCity = antHistories[(problemSize + 1) * i + 0];
            }
            else{
                endingCity = antHistories[(problemSize + 1) * i + j + 1];
            }

            // at the edge traveled, update the pheromone matrix according to the fitness of the ant's solution

            // get the amount to add to the edge (ant's total tour length is stored at the last index of the history)
            double pheromoneToAdd = 1/(antHistories[i * (problemSize+1) + problemSize]);

            pheromoneMatrix[problemSize * startingCity + endingCity] = pheromoneMatrix[problemSize * startingCity + endingCity] + pheromoneToAdd;
        }
    }

}

void ACOsolveSTSP(int problemSize, string location, int numAnts, int numIterations){
    // populate an adjacency matrix of the problem
    int adjacencyMatrixSize = sizeof(double) * problemSize * problemSize;
    double* adjacencyMatrix = (double*)malloc(adjacencyMatrixSize);


    getSTSPAdjacencyMatrix(adjacencyMatrix, location, problemSize);


    // create coppies of the problems on the device
    double* device_adjacencyMatrix;
    cudaHandleError(hipMalloc(&device_adjacencyMatrix, adjacencyMatrixSize));
    cudaHandleError(hipMemcpy(device_adjacencyMatrix, adjacencyMatrix, adjacencyMatrixSize, hipMemcpyHostToDevice));



    // allocate pheromone matrix on host and device
    int pheromoneMatrixSize = sizeof(double) * problemSize * problemSize;
    double* pheromoneMatrix = (double*)malloc(pheromoneMatrixSize);

    double* device_pheromoneMatrix;
    cudaHandleError(hipMalloc(&device_pheromoneMatrix, pheromoneMatrixSize));



    // allocate ant histories on matrix (with an additoinal comun to hold the distance traveled so far)
    int antHistoriesSize = sizeof(int) * numAnts * (problemSize + 1);
    int* antHistories = (int*)malloc(antHistoriesSize);

    int* device_antHistories;
    cudaHandleError(hipMalloc(&device_antHistories, antHistoriesSize));


    // for the given number of iterations
    for(int i = 0; i < numIterations; i++){
        


        // invoke kernel
        // check for kernel errors (immediately after kernel execution)

        // retrieve ant histories
        cudaHandleError(hipMemcpy(antHistories, device_antHistories, antHistoriesSize, hipMemcpyDeviceToHost));

        // update pheromone matrix
        updatePheromoneMatrix(pheromoneMatrix, pheromoneMatrixSize, antHistories, antHistoriesSize, numAnts, i, problemSize);
    }
    



    // get ant histories and find best result




    // free all used memory

        // device
    cudaHandleError(hipFree(device_adjacencyMatrix));
    cudaHandleError(hipFree(device_pheromoneMatrix));
    cudaHandleError(hipFree(device_antHistories));

        // host
    free(adjacencyMatrix);
    free(pheromoneMatrix);
    free(antHistories);
}

void ACOsolveATSP(int problemSize, string location, int numAnts, int numIterations, int nullKey){
    // populate an adjacency matrix of the problem
    int adjacencyMatrixSize = sizeof(int) * problemSize * problemSize;
    int* adjacencyMatrix = (int*)malloc(adjacencyMatrixSize);

    getATSPAdjacencyMatrix(adjacencyMatrix, location, nullKey);


    // create coppies of the problems on the device
    int* device_adjacencyMatrix;
    cudaHandleError(hipMalloc(&device_adjacencyMatrix, adjacencyMatrixSize));
    cudaHandleError(hipMemcpy(device_adjacencyMatrix, adjacencyMatrix, adjacencyMatrixSize, hipMemcpyHostToDevice));


    // allocate pheromone matrix on host and device
    int pheromoneMatrixSize = sizeof(double) * problemSize * problemSize;
    double* pheromoneMatrix = (double*)malloc(pheromoneMatrixSize);
    
    double* device_pheromoneMatrix;
    cudaHandleError(hipMalloc(&device_pheromoneMatrix, pheromoneMatrixSize));


    // allocate ant histories on matrix (with an additional column at the end to hold the total distance traveled so far)
    int antHistoriesSize = sizeof(int) * numAnts * (problemSize + 1);
    int* antHistories = (int*)malloc(antHistoriesSize);

    int* device_antHistories;
    cudaHandleError(hipMalloc(&device_antHistories, sizeof(int) * numAnts * problemSize));

    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result
    



    // free all used memory

        // device
    cudaHandleError(hipFree(device_adjacencyMatrix));
    cudaHandleError(hipFree(device_pheromoneMatrix));
    cudaHandleError(hipFree(device_antHistories));

        // host
    free(adjacencyMatrix);
    free(pheromoneMatrix);
    free(antHistories);

}


int main()
{

    // for a given problem size
    int STSPproblemSize = 1400; // number of cities
    int ATSPproblemSize = 65; // number of cites

    // and data in local file at a given location
    string STSPLocation = "fl1400.tsp";
    string ATSPLocation = "ftv64.atsp";

    // for a given number of ants
    int numAnts = 10000;

    // run a given number of iterations
    int numIterations = 1000;

    // and possibly some null key for data integrity
    int nullKey = 100000000;

    ACOsolveSTSP(STSPproblemSize, STSPLocation, numAnts, numIterations);
    ACOsolveATSP(ATSPproblemSize, ATSPLocation, numAnts, numIterations, nullKey);

    ACOPrint<<<GROUPS_OF_N_ANTS, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
    atexit(myexit);
    return EXIT_SUCCESS;
}


//
//__global__ void calculate_next_iteration(double** adj_mat, int* solutions, double* fitnesses, int solution_size);

//
//__global__ void print_info()
//{
//    unsigned int thread_id = threadIdx.x;
//    unsigned int block_id = blockIdx.x;
//    unsigned int index = block_id * blockDim.x + thread_id;
//    printf("Hello from thread %u on block %u. With a block dimension of %u, "
//           "this thread is operating on index %u\n",
//           thread_id, block_id, blockDim.x, index);
//}
//
//#define DEBUG false
//
//int main()
//{
//    /*
//    print_info<<<4, 32>>>();
//    hipDeviceSynchronize();
//    hipDeviceReset();
//
//    //these are not allocated on the heap, so we don't have to free
//    int test[4] = {1, 2, 3, 4};
//    float test1[5] = {1.00012512, 2.3, 9.0, 0.2 + 0.1, 1e7};
//
//    HelperFunctions::Host_PrintArray<int>(test, 4);
//    HelperFunctions::Host_PrintArray<float>(test1, 5, 16);
//
//    Node node_test[5] = {{40,50, 0}, {25, 85, 1}, {24, 29, 2}, {97, 2, 3}, {40, 50, 4}};
//    HelperFunctions::Host_PrintArray<Node>(node_test, 5);
//     */
//
//    int n_devices;
//    hipGetDeviceCount(&n_devices);
//    printf("Number of CUDA devices: %d\n", n_devices);
//    hipDeviceProp_t prop{};
//    cudaGetDeviceProperties_v2(&prop, 0);
//    printf("%s card information:\n", prop.name);
//    printf("\tMax threads per block:    \t%d\n", prop.maxThreadsPerBlock);
//    printf("\tMax threads per SM:       \t%d\n", prop.maxThreadsPerMultiProcessor);
//    printf("\tMax thread blocks per SM: \t%d\n", prop.maxBlocksPerMultiProcessor);
//    printf("\tMultiprocessor count:     \t%d\n", prop.multiProcessorCount);
//    printf("\tMax grid size:            \t%d\n", prop.maxGridSize[0]);
//    hipSetDevice(0);
//
//    static const int multiprocessors = prop.multiProcessorCount;
//    static const int threads_per_block = prop.maxThreadsPerBlock;
//
//    static const int test_numBlocks = 8;
//    static const int test_threadsPerBlock = 32;
//
//    int num_blocks = 0;
//    int num_threads = 0;
//    if(DEBUG)
//    {
//        num_blocks = test_numBlocks;
//        num_threads = test_threadsPerBlock;
//    }
//    else
//    {
//        num_blocks = multiprocessors;
//        num_threads = threads_per_block;
//    }
//
//    const int solution_size = 100;
//    int total_length = num_blocks * num_threads * solution_size;
//    int num_subsequences = total_length / solution_size;
//
//    std::cout << "Generating initial solutions..." << std::endl;
//
//    //create the repeating sequence 0, 1, 2, ..., solution_size - 1, 0, 1, 2, ...
//    thrust::counting_iterator<int> solutions_begin(0);
//    thrust::counting_iterator<int> solutions_end = solutions_begin + total_length;
//    thrust::device_vector<int> solutions(total_length);
//    thrust::transform(solutions_begin, solutions_end, solutions.begin(), [=]__device__(int x) { return x % solution_size; });
//
//    // Create a permutation vector to shuffle each subsequence
//    thrust::device_vector<int> permutation(solution_size);
//    thrust::sequence(permutation.begin(), permutation.end());
//
//    thrust::default_random_engine rng;
//    // Shuffle each subsequence independently
//    for (int i = 0; i < num_subsequences; ++i) {
//        int offset = i * solution_size;
//        thrust::device_vector<int>::iterator first = solutions.begin() + offset;
//        thrust::device_vector<int>::iterator last = first + solution_size;
//        thrust::shuffle(first, last, rng);
//    }
//
//    // Rearrange the elements of the sequence according to the shuffled permutation
//    thrust::device_vector<int> temp(total_length);
//    for (int i = 0; i < num_subsequences; ++i) {
//        int offset = i * solution_size;
//        thrust::device_vector<int>::iterator src_first = solutions.begin() + offset;
//        thrust::device_vector<int>::iterator src_last = src_first + solution_size;
//        thrust::device_vector<int>::iterator dst_first = temp.begin() + offset;
//        thrust::gather(permutation.begin(), permutation.end(), src_first, dst_first);
//    }
//    solutions = temp;
//    hipDeviceSynchronize();
//    int* device_solutions = thrust::raw_pointer_cast(solutions.data());
//
//    std::cout << "Initial solutions generated" << std::endl;
//
//    /*
//    int* host_solutions;
//    host_solutions = (int*) calloc(total_length, sizeof(int));
//    hipMemcpy(host_solutions, device_solutions, total_length * sizeof(int), hipMemcpyDeviceToHost);
//
//
//
//    std::cout << "here" << std::endl;
//    for(int i = total_length - solution_size; i < total_length; i++)
//    {
//        if(i % solution_size == 0 && i > 0) std::cout << std::endl;
//        std::cout << host_solutions[i] << ", ";
//
//    }
//    std::cout << std::endl;
//     */
//
//    calculate_next_iteration<<<num_blocks, num_threads>>>(nullptr, device_solutions, nullptr, solution_size);
//    hipDeviceSynchronize();
//
//
//    atexit(myexit);
//    return EXIT_SUCCESS;
//}
//
//__global__ void calculate_next_iteration(double** adj_mat, int* solutions, double* fitnesses, const int solution_size)
//{
//    auto thread_id = threadIdx.x;
//    auto block_id = blockIdx.x;
//    auto index = block_id * blockDim.x + thread_id;
//    auto start = index * solution_size;
//    auto end = start + solution_size;
//
//    //if(thread_id == 0 || thread_id == 1023)
//    if(index == 81919)
//    {
//        printf("Hello from thread %u on block %u aka index %u. I am operating on the solution solutions[%u, %u]. My fitness index is %u\n", thread_id, block_id, index, start, end - 1, index);
//        //unsigned long encode = 0;
//        for (auto i = start; i < end; ++i)
//        {
//            printf("%d ", solutions[i]);
//            //encode += static_cast<unsigned long>(pow(10, (solution_size - i + start - 1))) * solutions[i];
//            //printf("%d\n", (solution_size - i + start));
//            //printf("%f\n", pow(10, (solution_size - i + start)));
//        }
//
//        printf("\n");
//        //printf("%lu\n", encode);
//    }
//}
//
void myexit()
{
    printf("Exiting and resetting device\n");
    hipError_t err = hipDeviceReset();
    if(err != hipSuccess)
    {
        printf("Error: %s", hipGetErrorString(err));
    }
}